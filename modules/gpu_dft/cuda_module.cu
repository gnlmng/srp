#include "hip/hip_runtime.h"
// FFT functions (GPU)

#include "../../util/timer.h"

#include <vector>
#include <complex>
#include <cassert>
#include <hip/hip_complex.h>

extern bool MODULE_TIMED_OUT[];
extern size_t CURRENT_MODULE;

namespace gpu_dft {
	
	namespace _cuda__global__ {
		
		__global__ void dft(const size_t size, hipDoubleComplex* buffer, hipDoubleComplex* input) {
			__shared__ hipDoubleComplex cache[1024];
			
			cache[threadIdx.x] = make_hipDoubleComplex(0.0, 0.0);
			
			size_t i = blockIdx.y * gridDim.x + blockIdx.x;
			size_t j = threadIdx.x;
			
			if (i > size || j > size) {
				return;
			}
			
			hipDoubleComplex sum = make_hipDoubleComplex(0.0, 0.0);
			
			for (; j < size; j += 1024) {
				double angle = 2.0 * std::acos(-1.0) * (double) j * (double) i / (double) size;
				hipDoubleComplex ep = make_hipDoubleComplex(cos(angle), sin(angle));
				
				sum = hipCadd(sum, hipCmul(input[j], ep));
			}
			
			cache[threadIdx.x] = sum;
			
			__syncthreads();
			
			size_t k = threadIdx.x;
			
			for (size_t it = 1; it < 1024 && !(k & 1); it <<= 1, k >>= 1) {
				cache[threadIdx.x] = hipCadd(cache[threadIdx.x], cache[threadIdx.x + it]);
				__syncthreads();
			}
			
			if (!threadIdx.x) {
				buffer[blockIdx.y * gridDim.x + blockIdx.x] = cache[0];
			}
		}
		
	} /// namespace _cuda__global__
	
	timestamp _run(std::vector <std::complex <double>>& input) {
		if (input.size() > (size_t) 65536) {
			MODULE_TIMED_OUT[CURRENT_MODULE] = true;
			return (timestamp) 8'500'000;
		}
		
		static_assert(sizeof(hipDoubleComplex) == sizeof(std::complex <double>));
		
		const size_t size = input.size();
		
		hipDoubleComplex* in_buffer;
		hipDoubleComplex* out_buffer;
		
		hipMallocManaged(&in_buffer, sizeof(hipDoubleComplex) * size);
		hipMallocManaged(&out_buffer, sizeof(hipDoubleComplex) * size);
		
		hipMemcpy(in_buffer, input.data(), sizeof(std::complex <double>) * size, hipMemcpyHostToDevice);
		hipMemset(out_buffer, 0, sizeof(hipDoubleComplex) * size);
		
		Timer timer;
		
		size_t block_size = 1024;
		size_t grid_count = size;
		dim3 grid_size(grid_count);
		if (grid_count > 65535) {
			size_t value = ceil(sqrt(grid_count));
			grid_size = dim3(value, value);
		}
		
		_cuda__global__::dft <<<grid_size, block_size>>> (size, out_buffer, in_buffer);
		hipDeviceSynchronize();
		
		timestamp result_time = timer.current();
		
		hipMemcpy(input.data(), out_buffer, sizeof(std::complex <double>) * size, hipMemcpyDeviceToHost);
		
		return result_time;
	}
	
} /// namespace gpu_dft
