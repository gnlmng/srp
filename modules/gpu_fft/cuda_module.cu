#include "hip/hip_runtime.h"
// FFT functions (GPU)

#include "../../util/timer.h"

#include <vector>
#include <complex>
#include <cassert>
#include <hip/hip_complex.h>

namespace gpu_fft {
	
	namespace _cuda__global__ {
		
		__global__ void rearrange(const size_t size, hipDoubleComplex* buffer, size_t* reverse) {
			size_t x = blockIdx.x * blockDim.x + threadIdx.x;
			size_t y = blockIdx.y * blockDim.y + threadIdx.y;
			size_t sx = blockDim.x * gridDim.x;
			size_t index = y * sx + x;
			
			if (index >= size) {
				return;
			}
			
			if (index < reverse[index]) {
				auto tmp = buffer[index];
				buffer[index] = buffer[reverse[index]];
				buffer[reverse[index]] = tmp;
			}
		}
		
		__global__ void fft(const size_t size, hipDoubleComplex* buffer,
		hipDoubleComplex* roots, const size_t sub_size) {
			size_t x = blockIdx.x * blockDim.x + threadIdx.x;
			size_t y = blockIdx.y * blockDim.y + threadIdx.y;
			size_t sx = blockDim.x * gridDim.x;
			size_t index = y * sx + x;
			
			if (index >= (size >> 1)) {
				return;
			}
			
			size_t block = (index / sub_size) * (sub_size << 1);
			size_t offset = index % sub_size;
			
			hipDoubleComplex delta = hipCmul(roots[sub_size + offset], buffer[sub_size + block + offset]);
			
			buffer[sub_size + block + offset] = hipCsub(buffer[block + offset], delta);
			buffer[block + offset] = hipCadd(buffer[block + offset], delta);
		}
		
	} /// namespace _cuda__global__
	
	class FFT {
		
	public:
		
		FFT(const size_t _size) :
		m_size(_size)
		{
			assert(!_size || !(_size ^ (_size & ~(_size - 1))));
			
			hipMallocManaged(&m_roots, sizeof(hipDoubleComplex) * _size);
			hipMallocManaged(&m_reverse, sizeof(size_t) * _size);
			hipMallocManaged(&m_buffer, sizeof(hipDoubleComplex) * _size);
			
			m_roots[0] = make_hipDoubleComplex(1.0, 0.0);
			m_roots[1] = make_hipDoubleComplex(1.0, 0.0);
			
			for (size_t i = 2; i < _size; i <<= 1) {
				static const double pi = std::acos(-1.0);
				double inv = pi / (double) i;
				hipDoubleComplex angle = make_hipDoubleComplex(std::cos(inv), std::sin(inv));
				for (size_t j = i; j < (i << 1); j++) {
					m_roots[j] = hipCmul(m_roots[j >> 1], (j & 1) ? angle : make_hipDoubleComplex(1.0, 0.0));
				}
			}
			
			const int leading = 63 - __builtin_clzll(_size);
			for (size_t i = 0; i < _size; i++) {
				m_reverse[i] = (m_reverse[i >> 1] | ((i & 1) << leading)) >> 1;
			}
		}
		
		~FFT() {
			if (m_roots) {
				hipFree(m_roots);
			}
			if (m_reverse) {
				hipFree(m_reverse);
			}
			if (m_buffer) {
				hipFree(m_buffer);
			}
		}
		
		void set_memory(std::vector <std::complex <double>>& p) {
			hipMemcpy(m_buffer, p.data(), sizeof(std::complex <double>) * m_size, hipMemcpyHostToDevice);
		}
		
		void get_memory(std::vector <std::complex <double>>& p) {
			hipMemcpy(p.data(), m_buffer, sizeof(std::complex <double>) * m_size, hipMemcpyDeviceToHost);
		}
		
		timestamp fft() {
			Timer timer;
			
			static_assert(sizeof(hipDoubleComplex) == sizeof(std::complex <double>));
			
			size_t block_size = 1024;
			size_t grid_count = (m_size + block_size - 1) / block_size;
			dim3 grid_size(grid_count);
			if (grid_count > 65535) {
				size_t value = ceil(sqrt(grid_count));
				grid_size = dim3(value, value);
			}
			
			_cuda__global__::rearrange <<<grid_size, block_size>>> (m_size, m_buffer, m_reverse);
			hipDeviceSynchronize();
			
			grid_count = ((m_size >> 1) + block_size - 1) / block_size;
			grid_size = dim3(grid_count);
			if (grid_count > 65535) {
				size_t value = ceil(sqrt(grid_count));
				assert(value <= (size_t) 65535);
				grid_size = dim3(value, value);
			}
			
			for (size_t sub_size = 1; sub_size < m_size; sub_size <<= 1) {
				_cuda__global__::fft <<<grid_size, block_size>>> (m_size, m_buffer, m_roots, sub_size);
				hipDeviceSynchronize();
			}
			
			return timer.current();
		}
		
	private:
		
		size_t m_size;
		
		hipDoubleComplex* m_roots;
		size_t* m_reverse;
		
		hipDoubleComplex* m_buffer;
		
	};
	
	timestamp _run(std::vector <std::complex <double>>& input) {
		FFT fft(input.size());
		fft.set_memory(input);
		timestamp result_time = fft.fft();
		fft.get_memory(input);
		return result_time;
	}
	
} /// namespace gpu_fft
